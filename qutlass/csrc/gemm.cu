/*
 * Copyright (C) 2025 Roberto L. Castro (Roberto.LopezCastro@ist.ac.at). All Rights Reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *       http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <ATen/ATen.h>
#include <torch/types.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <hip/hip_runtime.h>

#ifndef QUTLASS_DISABLE_PYBIND
#include <torch/extension.h>
#endif

#include "cutlass/cutlass.h"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"
#include "cutlass/util/packed_stride.hpp"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/device/gemm.h"
#include "cutlass/util/reference/device/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/host/gett.hpp"
#include "cutlass/util/reference/host/tensor_norm.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/detail/sm100_blockscaled_layout.hpp"

#include <gemm.h>

using namespace cute;

template <typename MmaTileShape, typename ClusterShape, typename PerSmTileShape_MNK,
          typename ArchTag,
          typename ElementA, typename LayoutATag, int AlignmentA,
          typename ElementB, typename LayoutBTag, int AlignmentB>
struct FpGemm {
    using ElementD = cutlass::bfloat16_t;
    using ElementC = cutlass::bfloat16_t;
    using LayoutCTag = cutlass::layout::RowMajor;
    using LayoutDTag = cutlass::layout::RowMajor;
    static constexpr int AlignmentD = 128 / cutlass::sizeof_bits<ElementD>::value;
    static constexpr int AlignmentC = 128 / cutlass::sizeof_bits<ElementC>::value;

    using ElementAccumulator = float;
    using OperatorClass = cutlass::arch::OpClassBlockScaledTensorOp;

    using CollectiveEpilogue =
        typename cutlass::epilogue::collective::CollectiveBuilder<
            ArchTag, OperatorClass,
            PerSmTileShape_MNK, ClusterShape,
            cutlass::epilogue::collective::EpilogueTileAuto,
            ElementAccumulator, ElementAccumulator,
            ElementC, LayoutCTag, AlignmentC,
            ElementD, LayoutDTag, AlignmentD,
            cutlass::epilogue::collective::EpilogueScheduleAuto
            >::CollectiveOp;

    using CollectiveMainloop =
        typename cutlass::gemm::collective::CollectiveBuilder<
            ArchTag, OperatorClass,
            ElementA, LayoutATag, AlignmentA,
            ElementB, LayoutBTag, AlignmentB,
            ElementAccumulator,
            MmaTileShape, ClusterShape,
            cutlass::gemm::collective::StageCountAutoCarveout<
                static_cast<int>(
                    sizeof(typename CollectiveEpilogue::SharedStorage))>,
            cutlass::gemm::collective::KernelScheduleAuto
            >::CollectiveOp;

    using GemmKernel =
        cutlass::gemm::kernel::GemmUniversal<
            Shape<int, int, int, int>,
            CollectiveMainloop,
            CollectiveEpilogue,
            void>;

    using Gemm =
        cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
};

template <typename Gemm, typename ScaleType>
typename Gemm::Arguments args_from_options(
                                at::Tensor& D,
                                at::Tensor const& A,
                                at::Tensor const& B,
                                at::Tensor const& A_sf,
                                at::Tensor const& B_sf,
                                torch::Tensor const& alpha,
                                int M, int N, int K)
{
    using ElementA       = typename Gemm::ElementA;
    using ElementB       = typename Gemm::ElementB;
    using ElementD       = typename Gemm::ElementD;
    using ElementSFA     = ScaleType;
    using ElementSFB     = ScaleType;
    using ElementCompute = float;
    using ElementAccumulator = float;

    using StrideA = typename Gemm::GemmKernel::StrideA;
    using StrideB = typename Gemm::GemmKernel::StrideB;
    using StrideC = typename Gemm::GemmKernel::StrideC;
    using StrideD = typename Gemm::GemmKernel::StrideD;

    using Sm1xxBlkScaledConfig =
        typename Gemm::GemmKernel::CollectiveMainloop::Sm1xxBlkScaledConfig;

    auto stride_A = cutlass::make_cute_packed_stride(StrideA{}, {M, K, 1});
    auto stride_B = cutlass::make_cute_packed_stride(StrideB{}, {N, K, 1});
    auto stride_D = cutlass::make_cute_packed_stride(StrideD{}, {M, N, 1});

    auto layout_SFA = Sm1xxBlkScaledConfig::tile_atom_to_shape_SFA(
        cute::make_shape(M, N, K, 1));
    auto layout_SFB = Sm1xxBlkScaledConfig::tile_atom_to_shape_SFB(
        cute::make_shape(M, N, K, 1));

    typename Gemm::Arguments arguments{
        cutlass::gemm::GemmUniversalMode::kGemm,
        {M, N, K, 1},
        {
            static_cast<ElementA const*>(A.data_ptr()),      stride_A,
            static_cast<ElementB const*>(B.data_ptr()),      stride_B,
            static_cast<ElementSFA const*>(A_sf.data_ptr()), layout_SFA,
            static_cast<ElementSFB const*>(B_sf.data_ptr()), layout_SFB},
        {
            {},
            static_cast<ElementD const*>(D.data_ptr()), stride_D,
            static_cast<ElementD*>(D.data_ptr()),       stride_D
        }
    };
    auto& fusion_args = arguments.epilogue.thread;
    fusion_args.alpha_ptr = static_cast<ElementAccumulator const*>(alpha.data_ptr());

    return arguments;
}

template <typename Gemm, typename ScaleType>
void runGemm(at::Tensor& D,
             at::Tensor const& A,
             at::Tensor const& B,
             at::Tensor const& A_sf,
             at::Tensor const& B_sf,
             torch::Tensor const& alpha,
             int M, int N, int K,
             torch::Device device)
{
    Gemm gemm;

    auto arguments =
        args_from_options<Gemm, ScaleType>(D, A, B, A_sf, B_sf, alpha, M, N, K);

    size_t workspace_size = Gemm::get_workspace_size(arguments);

    cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

    const at::cuda::OptionalCUDAGuard device_guard(device_of(A));
    hipStream_t stream = at::cuda::getCurrentCUDAStream(device.index());

    CUTLASS_CHECK(gemm.can_implement(arguments));

    CUTLASS_CHECK(gemm.initialize(arguments, workspace.get(), stream));

    CUTLASS_CHECK(gemm.run(arguments, workspace.get(), stream));
}

void matmul_host_mxf4_bf16_tn(torch::Tensor& D,
                              torch::Tensor const& A,
                              torch::Tensor const& B,
                              torch::Tensor const& A_sf,
                              torch::Tensor const& B_sf,
                              torch::Tensor const& alpha)
{
    auto const m = A.sizes()[0];
    auto const n = B.sizes()[0];
    auto const k = A.sizes()[1] * 2;

    using ElementA   = cutlass::mx_float4_t<cutlass::float_e2m1_t>;
    using LayoutATag = cutlass::layout::RowMajor;
    static constexpr int AlignmentA = 128;

    using ElementB   = cutlass::mx_float4_t<cutlass::float_e2m1_t>;
    using LayoutBTag = cutlass::layout::ColumnMajor;
    static constexpr int AlignmentB = 128;

#if TARGET_CUDA_ARCH == 100 //TODO: improve tuning
    using ArchTag = cutlass::arch::Sm100;
    if(m<=16){
        using MmaTileShape       = Shape<_128,_128,_256>;
        using ClusterShape       = Shape<_1,_1,_1>;
        using PerSmTileShape_MNK = Shape<_128,_128,_256>;
        runGemm<FpGemm<MmaTileShape, ClusterShape, PerSmTileShape_MNK,
                        ArchTag,
                        ElementA, LayoutATag, AlignmentA,
                        ElementB, LayoutBTag, AlignmentB>::Gemm, cutlass::float_ue8m0_t
                    >(D, A, B, A_sf, B_sf, alpha, m, n, k, A.device());
    } else if(m<=256){
        using MmaTileShape       = Shape<_256,_128,_256>;
        using ClusterShape       = Shape<_2,_1,_1>;
        using PerSmTileShape_MNK = Shape<_128,_128,_256>;
        runGemm<FpGemm<MmaTileShape, ClusterShape, PerSmTileShape_MNK,
                        ArchTag,
                        ElementA, LayoutATag, AlignmentA,
                        ElementB, LayoutBTag, AlignmentB>::Gemm, cutlass::float_ue8m0_t
                    >(D, A, B, A_sf, B_sf, alpha, m, n, k, A.device());
    } else {
        using MmaTileShape       = Shape<_256,_256,_256>;
        using ClusterShape       = Shape<_2,_1,_1>;
        using PerSmTileShape_MNK = Shape<_128,_256,_256>;
        runGemm<FpGemm<MmaTileShape, ClusterShape, PerSmTileShape_MNK,
                        ArchTag,
                        ElementA, LayoutATag, AlignmentA,
                        ElementB, LayoutBTag, AlignmentB>::Gemm, cutlass::float_ue8m0_t
                    >(D, A, B, A_sf, B_sf, alpha, m, n, k, A.device());
    }
#elif TARGET_CUDA_ARCH == 120
    using ArchTag = cutlass::arch::Sm120;
    using ClusterShape       = Shape<_1,_1,_1>;
    if(m<512){
        using MmaTileShape       = Shape<_128,_128,_128>;
        using PerSmTileShape_MNK = Shape<_128,_128,_128>;

        runGemm<FpGemm<MmaTileShape, ClusterShape, PerSmTileShape_MNK,
                        ArchTag,
                        ElementA, LayoutATag, AlignmentA,
                        ElementB, LayoutBTag, AlignmentB>::Gemm, cutlass::float_ue8m0_t
                >(D, A, B, A_sf, B_sf, alpha, m, n, k, A.device());
    } else {
        using MmaTileShape       = Shape<_256,_128,_128>;
        using PerSmTileShape_MNK = Shape<_256,_128,_128>;

        runGemm<FpGemm<MmaTileShape, ClusterShape, PerSmTileShape_MNK,
                        ArchTag,
                        ElementA, LayoutATag, AlignmentA,
                        ElementB, LayoutBTag, AlignmentB>::Gemm, cutlass::float_ue8m0_t
                >(D, A, B, A_sf, B_sf, alpha, m, n, k, A.device());
    }
#else
    TORCH_CHECK(false, "Unsupported CUDA arch");
#endif
}

void matmul_host_nvf4_bf16_tn(torch::Tensor& D,
                              torch::Tensor const& A,
                              torch::Tensor const& B,
                              torch::Tensor const& A_sf,
                              torch::Tensor const& B_sf,
                              torch::Tensor const& alpha)
{
    auto const m = A.sizes()[0];
    auto const n = B.sizes()[0];
    auto const k = A.sizes()[1] * 2;

    using ElementA   = cutlass::nv_float4_t<cutlass::float_e2m1_t>;
    using LayoutATag = cutlass::layout::RowMajor;
    static constexpr int AlignmentA = 32;

    using ElementB   = cutlass::nv_float4_t<cutlass::float_e2m1_t>;
    using LayoutBTag = cutlass::layout::ColumnMajor;
    static constexpr int AlignmentB = 32;

#if TARGET_CUDA_ARCH == 100 //TODO: improve tuning
    using ArchTag = cutlass::arch::Sm100;
    if(m<=16){
        using MmaTileShape       = Shape<_128,_128,_256>;
        using ClusterShape       = Shape<_1,_1,_1>;
        using PerSmTileShape_MNK = Shape<_128,_128,_256>;
        runGemm<FpGemm<MmaTileShape, ClusterShape, PerSmTileShape_MNK,
                        ArchTag,
                        ElementA, LayoutATag, AlignmentA,
                        ElementB, LayoutBTag, AlignmentB>::Gemm, cutlass::float_ue4m3_t
                    >(D, A, B, A_sf, B_sf, alpha, m, n, k, A.device());
    } else if(m<=256){
        using MmaTileShape       = Shape<_256,_128,_256>;
        using ClusterShape       = Shape<_2,_1,_1>;
        using PerSmTileShape_MNK = Shape<_128,_128,_256>;
        runGemm<FpGemm<MmaTileShape, ClusterShape, PerSmTileShape_MNK,
                        ArchTag,
                        ElementA, LayoutATag, AlignmentA,
                        ElementB, LayoutBTag, AlignmentB>::Gemm, cutlass::float_ue4m3_t
                    >(D, A, B, A_sf, B_sf, alpha, m, n, k, A.device());
    } else {
        using MmaTileShape       = Shape<_256,_256,_256>;
        using ClusterShape       = Shape<_2,_1,_1>;
        using PerSmTileShape_MNK = Shape<_128,_256,_256>;
        runGemm<FpGemm<MmaTileShape, ClusterShape, PerSmTileShape_MNK,
                        ArchTag,
                        ElementA, LayoutATag, AlignmentA,
                        ElementB, LayoutBTag, AlignmentB>::Gemm, cutlass::float_ue4m3_t
                    >(D, A, B, A_sf, B_sf, alpha, m, n, k, A.device());
    }
#elif TARGET_CUDA_ARCH == 120
    using ArchTag = cutlass::arch::Sm120;
    using ClusterShape       = Shape<_1,_1,_1>;

    if(m<512){
        using MmaTileShape       = Shape<_128,_128,_128>;
        using PerSmTileShape_MNK = Shape<_128,_128,_128>;

        runGemm<FpGemm<MmaTileShape, ClusterShape, PerSmTileShape_MNK,
                        ArchTag,
                        ElementA, LayoutATag, AlignmentA,
                        ElementB, LayoutBTag, AlignmentB>::Gemm, cutlass::float_ue4m3_t
                >(D, A, B, A_sf, B_sf, alpha, m, n, k, A.device());
    } else {
        using MmaTileShape       = Shape<_256,_128,_128>;
        using PerSmTileShape_MNK = Shape<_256,_128,_128>;

        runGemm<FpGemm<MmaTileShape, ClusterShape, PerSmTileShape_MNK,
                        ArchTag,
                        ElementA, LayoutATag, AlignmentA,
                        ElementB, LayoutBTag, AlignmentB>::Gemm, cutlass::float_ue4m3_t
                >(D, A, B, A_sf, B_sf, alpha, m, n, k, A.device());
    }
#else
    TORCH_CHECK(false, "Unsupported CUDA arch");
#endif

}

void matmul_host_mxf8_bf16_tn(torch::Tensor& D,
                              torch::Tensor const& A,
                              torch::Tensor const& B,
                              torch::Tensor const& A_sf,
                              torch::Tensor const& B_sf,
                              torch::Tensor const& alpha)
{
    auto const m = A.sizes()[0];
    auto const n = B.sizes()[0];
    auto const k = A.sizes()[1];

    using ElementA   = cutlass::mx_float8_t<cutlass::float_e4m3_t>;
    using LayoutATag = cutlass::layout::RowMajor;
    static constexpr int AlignmentA = 16;

    using ElementB   = cutlass::mx_float8_t<cutlass::float_e4m3_t>;
    using LayoutBTag = cutlass::layout::ColumnMajor;
    static constexpr int AlignmentB = 16;

#if TARGET_CUDA_ARCH == 100
    using ArchTag = cutlass::arch::Sm100;

    if(m<=8192){
        using MmaTileShape       = Shape<_256,_128,_128>;
        using ClusterShape       = Shape<_2,_1,_1>;
        using PerSmTileShape_MNK = Shape<_128,_128,_128>;

        runGemm<FpGemm<MmaTileShape, ClusterShape, PerSmTileShape_MNK,
                        ArchTag,
                        ElementA, LayoutATag, AlignmentA,
                        ElementB, LayoutBTag, AlignmentB>::Gemm, cutlass::float_ue8m0_t
                    >(D, A, B, A_sf, B_sf, alpha, m, n, k, A.device());
    } else {
        using MmaTileShape       = Shape<_256,_256,_128>;
        using ClusterShape       = Shape<_2,_1,_1>;
        using PerSmTileShape_MNK = Shape<_128,_256,_128>;

        runGemm<FpGemm<MmaTileShape, ClusterShape, PerSmTileShape_MNK,
                        ArchTag,
                        ElementA, LayoutATag, AlignmentA,
                        ElementB, LayoutBTag, AlignmentB>::Gemm, cutlass::float_ue8m0_t
                    >(D, A, B, A_sf, B_sf, alpha, m, n, k, A.device());
    }
#elif TARGET_CUDA_ARCH == 120
    using ArchTag = cutlass::arch::Sm120;
    using MmaTileShape       = Shape<_128,_128,_128>;
    using ClusterShape       = Shape<_1,_1,_1>;
    using PerSmTileShape_MNK = Shape<_128,_128,_128>;

    runGemm<FpGemm<MmaTileShape, ClusterShape, PerSmTileShape_MNK,
                    ArchTag,
                    ElementA, LayoutATag, AlignmentA,
                    ElementB, LayoutBTag, AlignmentB>::Gemm, cutlass::float_ue8m0_t
                >(D, A, B, A_sf, B_sf, alpha, m, n, k, A.device());
#else
    TORCH_CHECK(false, "Unsupported CUDA arch");
#endif
}